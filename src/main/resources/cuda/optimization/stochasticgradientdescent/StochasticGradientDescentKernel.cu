
#include <hip/hip_runtime.h>
extern "C"
__global__ void stochasticGradientDescentKernel (
    int numberIterations,
    float learningRate,
    int* parameterIndices,
    int parameterSize,
    float* parameters,
    float scalingFactor,
    float* gradient)
{

    int startEntry = threadIdx.x * numberIterations;

    if(startEntry < parameterSize) {

        int indexParameter = parameterIndices[blockIdx.x];

        int startGradient = blockIdx.x * parameterSize + startEntry;
        int startParameter = indexParameter * parameterSize + startEntry;

        for(int i = 0; i < numberIterations; i++) {

            parameters[startParameter + i] -= scalingFactor * learningRate * gradient[startGradient + i];

        }

    }

}